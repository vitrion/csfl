extern const unsigned int samples(31);
#include "fls.cuh"
#include <fstream>

int main()
{
	bool isCUDA;
	bool isStream;
	vector<float> numOfInputs(512, 1.0);
	unsigned int numOfRules = 120;
	float sigma = 0.25;
	unsigned int numOfSetsPerVar = 4;

	// FLS initialization
	CUDAinit();

	cout << "T1FLS configuration started..." << endl;
	fls macro;
	macro.setName("macroTest");
	macro.setInferenceModel("Mamdani");	

	vector<float> range = { -1, 1 };
	vector<string> vnames;
	for (unsigned int i = 0; i < numOfInputs.size(); i++) {
		ostringstream convert;
		convert << i;
		string str = convert.str();
		vnames.push_back("x" + str);
		macro.addFuzzyVar("Input", vnames[i], range[0], range[1]);
	}
	macro.addFuzzyVar("Output", "y", -10, 10);

	// Generate automatically every set in every variable
	float dx = (range[1] - range[0]) / float(numOfSetsPerVar - 1);
	vector<float> r(numOfSetsPerVar + 2);
	for (unsigned int i = 0; i < numOfSetsPerVar + 2; i++)
		r[i] = float(i) * dx + range[0] - dx;
	for (unsigned int i = 0; i < macro.getInVarCount(); i++) {
		string varName = macro.getInVarName(i);
		ostringstream convert1;
		convert1 << i;
		string varID = convert1.str();
		for (unsigned int j = 0; j < numOfSetsPerVar; j++) {
			ostringstream convert2;
			convert2 << j;
			string setID = convert2.str();
			if (j == 0)
				macro.addFuzzySet(varName, "A" + varID + setID, "Z", { r[j + 1], r[j + 2] }, 1);
			else if (j == numOfSetsPerVar - 1)
				macro.addFuzzySet(varName, "A" + varID + setID, "S", { r[j], r[j + 1] }, 1);
			else
				macro.addFuzzySet(varName, "A" + varID + setID, "Triangular", { r[j], r[j + 1], r[j + 2] }, 1);
		}
	}
	string varName = macro.getOutVarName(0);
	for (unsigned int j = 0; j < numOfSetsPerVar; j++) {
		ostringstream convert2;
		convert2 << j;
		string setID = convert2.str();
		if (j == 0)
			macro.addFuzzySet(varName, "B" + setID, "Z", { r[j + 1], r[j + 2] }, 1);
		else if (j == numOfSetsPerVar - 1)
			macro.addFuzzySet(varName, "B" + setID, "S", { r[j], r[j + 1] }, 1);
		else
			macro.addFuzzySet(varName, "B" + setID, "Triangular", { r[j], r[j + 1], r[j + 2] }, 1);
	}
	// Generates all the possible rules according to the available premises
	macro.addFuzzyRule(numOfRules);
	// Generate the inference schedule before execution
	macro.configure();

	for (unsigned int h = 0; h < 1; h++) {			
		// Starts initialization
		if (h == 0){
			isCUDA = true;// false;
			isStream = true;// false;
		}
		else if (h == 1) {
			isCUDA = true;
			isStream = false;
		}
		else {
			isCUDA = true;
			isStream = true;
		}
		macro.setHetProc(isCUDA);
		macro.setStreams(isStream);

		vector<fs> x_primes(numOfInputs.size());
		for (unsigned int i = 0; i < numOfInputs.size(); i++) {
			ostringstream convert;
			convert << i;
			string str = convert.str();
			fs prime(range[0], range[1], 1.0, "x" + str + "_prime", "Gaussian", { numOfInputs[i], sigma }, isCUDA, isStream);
			x_primes[i] = prime;
		}

		// Ends initialization
		if (h == 0)
			cout << "Heterogeneous with Streams fuzzy processing started..." << endl;
		else if (h == 1)
			cout << "Heterogeneous fuzzy processing started..." << endl;
		else
			cout << "Heterogeneous with Streams fuzzy processing started..." << endl;
		// Starts execution

		// Fuzzification process
		macro.fuzzify(x_primes);
		// Save the rules back to a file to verify nested operations
		macro.saveResultingRulesAsStr("test2.rs");
		// Execute inference process according to inference schedule
		macro.infer();		
		// Defuzzification process
		vector<float> result = macro.defuzzify();

		// Ends execution
		if (h == 0){
			cout << "Heterogeneous with Streams fuzzy processing finished." << endl;
		else if (h == 1)
			cout << "Heterogeneous fuzzy processing finished." << endl;
		else
			cout << "Heterogeneous with Streams fuzzy processing finished." << endl;
		cout << result[0] << endl;
	}
	CUDAend();
    return 0;
}
